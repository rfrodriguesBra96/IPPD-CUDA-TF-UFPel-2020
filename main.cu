#include "hip/hip_runtime.h"


#include "nbody.hpp"
#include <cstdio>
#include <iostream>
#include <cmath>
#include <cstring>
#include <sys/time.h>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include "stdlib.h"
#include <cassert>
#include <ctime>

using namespace std;

__global__ void nbody(Particle* d_particles, Particle *output,int number_of_particles,float time_interval){
    
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(id < number_of_particles) {  
        
    Particle* this_particle = &output[id];
    
    float force_x = 0.0f, force_y = 0.0f, force_z = 0.0f;
    float total_force_x = 0.0f, total_force_y = 0.0f, total_force_z = 0.0f;
    int i;

    for(i = 0; i < number_of_particles; i++) {
                
      if(i != id) {
                                             
        Particle* this_particle1 = d_particles + id;
        Particle* this_particle2 = d_particles + i;
        float* force_x_aux = &force_x; 
        float* force_y_aux = &force_y; 
        float* force_z_aux = &force_z;                          

        float difference_x, difference_y, difference_z;
        float distance_squared, distance;
        float force_magnitude;

        difference_x = this_particle2->position_x - this_particle1->position_x;
        difference_y = this_particle2->position_y - this_particle1->position_y;
        difference_z = this_particle2->position_z - this_particle1->position_z;

        distance_squared = difference_x * difference_x +
                           difference_y * difference_y +
                           difference_z * difference_z;

        distance = std::sqrt(distance_squared);//sqrtf(distance_squared);

        force_magnitude = GRAVITATIONAL_CONSTANT * (this_particle1->mass) * (this_particle2->mass) / distance_squared;

        *force_x_aux = (force_magnitude / distance) * difference_x;
        *force_y_aux = (force_magnitude / distance) * difference_y;
        *force_z_aux = (force_magnitude / distance) * difference_z;
        

        total_force_x += force_x;
        total_force_y += force_y;
        total_force_z += force_z;
      }                        
    }
       
        float velocity_change_x, velocity_change_y, velocity_change_z;
        float position_change_x, position_change_y, position_change_z;
        
        this_particle->mass = d_particles[id].mass;
            
        velocity_change_x = total_force_x * (time_interval / this_particle->mass);
        velocity_change_y = total_force_y * (time_interval / this_particle->mass);
        velocity_change_z = total_force_z * (time_interval / this_particle->mass);
            
        position_change_x = d_particles[id].velocity_x + velocity_change_x * (0.5 * time_interval);
        position_change_y = d_particles[id].velocity_y + velocity_change_y * (0.5 * time_interval);
        position_change_z = d_particles[id].velocity_z + velocity_change_z * (0.5 * time_interval);
            
        this_particle->velocity_x = d_particles[id].velocity_x + velocity_change_x;
        this_particle->velocity_y = d_particles[id].velocity_y + velocity_change_y;
        this_particle->velocity_z = d_particles[id].velocity_z + velocity_change_z;
        
        this_particle->position_x = d_particles[id].position_x + position_change_x;
        this_particle->position_y = d_particles[id].position_y + position_change_y;
        this_particle->position_z = d_particles[id].position_z + position_change_z;

  }

}

int main (int argc, char** argv) { 
    if(argc < 2) {
        std::cout << "Informe um arquivo com os parâmetros de entrada: ./nbody_simulation <input_file.in>\n";
        std::abort();
    }
   int n = atoi(argv[2]);

    
    Particle* particle_array  = nullptr;
    Particle* particle_array2 = nullptr;
    Particle* dev_particle_array;
    Particle* dev_particle_array2; 
    
    FILE *input_data = fopen(argv[1], "r");
    Particle_input_arguments(input_data);

    particle_array  = Particle_array_construct(number_of_particles);
  particle_array2 = Particle_array_construct(number_of_particles);

    Particle_array_initialize(particle_array, number_of_particles);
    
  printf("\nProcessando simulação NBody....\n");
    
    
  long start = wtime();   
    
  hipMalloc((void**)&dev_particle_array, number_of_particles * sizeof(Particle));
  hipMalloc((void**)&dev_particle_array2, number_of_particles * sizeof(Particle));
    
  hipMemcpy(dev_particle_array, particle_array, number_of_particles *sizeof(Particle),hipMemcpyHostToDevice);
  hipMemcpy(dev_particle_array2, particle_array2, number_of_particles *sizeof(Particle),hipMemcpyHostToDevice);
   
  //!-------KERNEL-----
  
  for(int timestep = 1; timestep <= number_of_timesteps; timestep++) {
     
    nbody<<<number_of_particles,n>>>(dev_particle_array,dev_particle_array2,number_of_particles,time_interval);
        
    /* swap arrays */
    Particle * tmp = particle_array;
    particle_array = particle_array2;
    particle_array2 = tmp;
        
    //printf("   Iteração %d OK\n", timestep);
        
    hipError_t err = hipMemcpy(particle_array, dev_particle_array, number_of_particles * sizeof(Particle), hipMemcpyDeviceToHost);
    if(err != hipSuccess) {
      printf("CUDA error ao copiar dados para o Host: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
   printf(" iteração: %d OK\n ",timestep);
  }
        
  hipFree(dev_particle_array);
  hipFree(dev_particle_array2);

    long end = wtime();
    double time = (end - start) / 1000000.0;

   
    printf("\nSimulação NBody executada com sucesso.\n");
    
    printf("Numero de threads: %d\n",n);
    printf("Nro. de Iterações : %d\n", number_of_timesteps);
    printf("Nro. de Partículas: %d\n", number_of_particles);
    printf("Tempo: %.8f segundos\n", time);


    // #ifdef VERBOSE
        //Imprimir saída para arquivo
        printf("\nImprimindo saída em arquivo...\n");
        FILE * fileptr = fopen("nbody_simulation.out", "w");
        Particle_array_output_xyz(fileptr, particle_array, number_of_particles);
        printf("Saída da simulação salva no arquivo nbody_simulation.out\n");
    // #endif
                 
        particle_array  = Particle_array_destruct(particle_array, number_of_particles);
        particle_array2 = Particle_array_destruct(particle_array2, number_of_particles);
  
      
    
    return PROGRAM_SUCCESS_CODE;
  }